/* SPDX-License-Identifier: Apache-2.0
 *
 * Copyright 2023 Damian Peckett <damian@peckett>.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <chrono>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void add(int32_t n, int32_t *x, int32_t *y) {
  auto index = threadIdx.x;
  auto stride = blockDim.x;

  for (auto i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main() {
  auto N = 1 << 20; // 1M elements
  int32_t *x, *y;

  std::cout << "Allocating memory" << std::endl;

  hipMallocManaged(&x, N * sizeof(int32_t));
  hipMallocManaged(&y, N * sizeof(int32_t));

  // initialize x and y arrays on the host
  for (auto i = 0; i < N; i++) {
    x[i] = 1;
    y[i] = 2;
  }

  auto start = std::chrono::high_resolution_clock::now();

  std::cout << "Beginning stress test" << std::endl;

  // Run kernel on 1M elements on the GPU
  int64_t num_iterations = 100000000LL;
  for (auto i = 0; i < num_iterations; ++i) {
    add<<<1, 256>>>(N, x, y);
    hipDeviceSynchronize();

    // Print time elapsed every few seconds
    if (i > 0 && i % 1000 == 0) {
      auto end = std::chrono::high_resolution_clock::now();
      std::chrono::duration<double> elapsed = end - start;
      start = end;

      std::cout << "Iterations per second: " << (1000.0 / elapsed.count())
                << std::endl;
    }
  }

  hipFree(x);
  hipFree(y);

  return 0;
}
